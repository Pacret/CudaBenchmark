#include "hip/hip_runtime.h"
#include "includes.hpp"
#include "Bench/CPU/CpuBinarize.hpp"
#include "Bench/CPU/CpuSobel.hpp"
#include "Bench/CPU/CpuErosion.hpp"
#include "Bench/GPU/Binarize/GpuBinarize.hpp"
#include "Bench/GPU/Binarize/GpuBinarizeCopy.hpp"
#include "Bench/GPU/Binarize/GpuBinarizePrefetch.hpp"
#include "Bench/GPU/Binarize/GpuBinarizeMemcopy.hpp"
#include "Bench/GPU/Binarize/GpuBinarizeMemcopyNotCount.hpp"
#include "Bench/GPU/Sobel/GpuSobel.hpp"
#include "Bench/GPU/Sobel/GpuSobelPrefetch.hpp"
#include "Bench/GPU/Sobel/GpuSobelMemcopy.hpp"
#include "Bench/GPU/Sobel/GpuSobelMemcopyNotCount.hpp"


void        displayConsole(std::vector<Bench *> benchs)
{
    for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
        (*it)->benchmark();

    std::sort(benchs.begin(), benchs.end(), [](Bench *r1, Bench *r2) -> bool {
                                          return r1->duration8bits > r2->duration8bits; });
   for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
    {
        int max = min((int)(*it)->times8bits.size(), 10);
        std::cout << (*it)->times8bits[0] << std::endl;
        std::cout << (*it)->name << " (" << (*it)->xSize << ", " << (*it)->ySize << ") : " << std::endl;
        std::cout << "      8 bits : " << (*it)->duration8bits << " ms      (";
        for (int i = 0; i < max; i++)
        {
            std::cout << (*it)->times8bits[i];
            if (i != max - 1)
                std::cout << ", ";
        }
        std::cout << ")" << std::endl;
        std::cout << "     16 bits : " << (*it)->duration16bits << " ms     (";
        for (int i = 0; i < max; i++)
        {
            std::cout << (*it)->times16bits[i];
            if (i != max - 1)
                std::cout << ", ";
        }
        std::cout << ")" << std::endl;
    }
}

void        displayCsv(std::vector<Bench *> benchs, int c)
{
    for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
        (*it)->benchmark();

    int i = 0;
    std::vector<std::string> lines;
    for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
    {

        if (i < c)
        {
            lines.push_back((*it)->name + " 8 bits, " + std::to_string((*it)->duration8bits) + ", ");
        }
        else
            lines[i % c] += std::to_string((*it)->duration8bits) + ", ";
        i++;
    }
    for (auto it = lines.crbegin(); it != lines.crend(); ++it)
    {
        std::cout << *(it) << std::endl;
    }
    i = 0;
    lines.clear();
    for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
    {
        if (i < c)
        {
            lines.push_back((*it)->name + " 16 bits, " + std::to_string((*it)->duration16bits) + ", ");
        }
        else
        {
            lines[i % c] += std::to_string((*it)->duration16bits) + ", ";
        }
        i++;
    }
    for (auto it = lines.crbegin(); it != lines.crend(); ++it)
    {
        std::cout << *(it) << std::endl;
    }
}

int main()
{
    std::cout << "Start benchmark" << std::endl;
    // int xSize = 2048;
    // int ySize = 32;

    std::vector<Bench *> benchs;

    for (int i = 64; i <= 4096; i+=64)
    {
        std::cout << i << " x " << i << " = " << i*i << ", ";
        benchs.push_back(new GpuBinarizeMemcopyNotCount("GPU Binarize memcopy not count", i, i));
        benchs.push_back(new GpuBinarizePrefetch("GPU Binarize prefetch", i, i));
        benchs.push_back(new GpuBinarize("GPU Binarize", i, i));
        benchs.push_back(new CpuBinarize("CPU Binarize", i, i));
        benchs.push_back(new GpuSobel("GPU Sobel", i, i));
        benchs.push_back(new GpuSobelPrefetch("GPU Sobel prefetch", i, i));
        benchs.push_back(new GpuSobelMemcopy("GPU Sobel memcopy ", i, i));
        benchs.push_back(new GpuSobelMemcopyNotCount("GPU Sobel memcopy not count", i, i));
        benchs.push_back(new CpuSobel("CPU Sobel", i, i));
    }
    std::cout << std::endl;


    // for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
    // {
    //     int imageSize = (*(it))->xSize * (*(it))->ySize;
    //     auto before = system_clock::now();
    //     for (int i = 0; i < imageSize; i++)
    //     {
    //         (*(it))->image8bits[i] = rand() % 256;
    //     }
    //     millisec t = std::chrono::system_clock::now() - before;
    //     std::cout << "acces to value " << (*it)->name << " after calcul take : " << t.count() << " ms" << std::endl;
    // }


    // benchs.push_back(new CpuErosion("CPU Erosion", xSize, ySize));

    displayCsv(benchs, 9);
    //displayConsole(benchs);

    for (auto it = benchs.crbegin(); it != benchs.crend(); ++it)
        delete *it;
    return 0;
}