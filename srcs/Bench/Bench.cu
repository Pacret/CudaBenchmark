#include "Bench.hpp"

Bench::Bench(std::string name, int xSize, int ySize) : name(name), xSize(xSize), ySize(ySize)
{
    hipMallocManaged(&image8bits, sizeof(unsigned char) * (xSize * ySize));
    hipMallocManaged(&image16bits, sizeof(short) * (xSize * ySize));
}


Bench::~Bench()
{
    hipFree(image8bits);
    hipFree(image16bits);
}

void    Bench::randomizeImage()
{
    int imageSize = xSize * ySize;
    srand(time(NULL));
    for (int i = 0; i < imageSize; i++)
    {
        image8bits[i] = rand() % 256;
        image16bits[i] = rand() % 65536;
    }
}

bool    Bench::verifyImageValues()
{
    // for (size_t y = 0; y < ySize; y++)
    // {
    //     for (size_t x = 0; x < xSize; x++)
    //     {
    //         int index = y * xSize + x;
    //         if ((image8bits[index] != BINARIZE_SEUIL_BAS && image8bits[index] != BINARIZE_SEUIL_HAUT)
    //         || (image16bits[index] != BINARIZE_SEUIL_BAS && image16bits[index] != BINARIZE_SEUIL_HAUT))
    //         {

    //             if (x > 0 && x < xSize - 2 && y > 0 && y < ySize - 2)
    //             {
    //                 std::cout << "pas bon a " << x << ", " << y << std::endl;
    //                 if (image8bits[index] != SOBEL_GREY_VALUE || image16bits[index] != SOBEL_GREY_VALUE)
    //                     return false;
    //             }
    //         }

    //     }
    // }
    return true;
}

void    Bench::benchmark()
{
    for (int i = 0; i < N_BENCHMARK; i++)
    {
        randomizeImage();

        auto before8bits = system_clock::now();
        double toSubstract = process8bits(); 

        millisec t8bits = std::chrono::system_clock::now() - before8bits;
        double c = (t8bits.count()) - toSubstract;
        times8bits.push_back(c);
        duration8bits += c;

        auto before16bits = system_clock::now();
        toSubstract = process16bits();

        millisec t16bits = std::chrono::system_clock::now() - before16bits;
        c = (t16bits.count()) - toSubstract;
        times16bits.push_back(c);
        duration16bits += c;

    }
    bool noError = verifyImageValues();
    if (!noError)
        std::cout << name <<  " datas not correct" << std::endl;
    duration8bits -= times8bits[0];
    duration8bits /= (N_BENCHMARK - 1);

    duration16bits -= times16bits[0];
    duration16bits /= (N_BENCHMARK - 1);
}