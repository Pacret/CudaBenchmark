#include "GpuSobelPrefetch.hpp"

GpuSobelPrefetch::GpuSobelPrefetch(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuSobelPrefetch::~GpuSobelPrefetch()  
{
}


double    GpuSobelPrefetch::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    int device = -1;
    hipGetDevice(&device);

    hipMemPrefetchAsync(image8bits, xSize * ySize * sizeof(unsigned char), device, NULL);
    gpu_sobel8<<<gridSize, blockSize>>>(image8bits, xSize, ySize);
    hipDeviceSynchronize();
    return 0;
}

double    GpuSobelPrefetch::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(image16bits, xSize * ySize * sizeof(short), device, NULL);
    gpu_sobel16<<<gridSize, blockSize>>>(image16bits, xSize, ySize);
    hipDeviceSynchronize();
    return 0;
}