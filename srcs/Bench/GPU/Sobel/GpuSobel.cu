#include "GpuSobel.hpp"

GpuSobel::GpuSobel(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuSobel::~GpuSobel()  
{
}

double    GpuSobel::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    gpu_sobel8<<<gridSize, blockSize>>>(image8bits, xSize, ySize);
    hipDeviceSynchronize();
    return 0;
}


double    GpuSobel::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    gpu_sobel16<<<gridSize, blockSize>>>(image16bits, xSize, ySize);
    hipDeviceSynchronize();
    return 0;
}


