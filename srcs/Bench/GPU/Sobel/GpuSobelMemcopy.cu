#include "GpuSobelMemcopy.hpp"

GpuSobelMemcopy::GpuSobelMemcopy(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuSobelMemcopy::~GpuSobelMemcopy()  
{
}


double    GpuSobelMemcopy::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    

    unsigned char *image8bitsCOPY;
    
    hipMallocManaged(&image8bitsCOPY, sizeof(unsigned char) * (xSize * ySize));
    hipMemcpy(image8bits, image8bitsCOPY, sizeof(unsigned char) * xSize * ySize, hipMemcpyDefault); 

    gpu_sobel8<<<gridSize, blockSize>>>(image8bitsCOPY, xSize, ySize);
    hipDeviceSynchronize();
    hipFree(image8bitsCOPY);
    return 0;
}

double    GpuSobelMemcopy::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);

    short *image16bitsCOPY;
    hipMallocManaged(&image16bitsCOPY, sizeof(short) * (xSize * ySize));
    hipMemcpy(image16bits, image16bitsCOPY, sizeof(short) * xSize * ySize, hipMemcpyDefault); 

    gpu_sobel16<<<gridSize, blockSize>>>(image16bitsCOPY, xSize, ySize);

    hipDeviceSynchronize();
    hipFree(image16bitsCOPY);
    return 0;
}