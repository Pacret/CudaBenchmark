#include "GpuSobelMemcopyNotCount.hpp"

GpuSobelMemcopyNotCount::GpuSobelMemcopyNotCount(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuSobelMemcopyNotCount::~GpuSobelMemcopyNotCount()  
{
}


double    GpuSobelMemcopyNotCount::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    

    auto before = system_clock::now();
    unsigned char *image8bitsCOPY;
    
    hipMallocManaged(&image8bitsCOPY, sizeof(unsigned char) * (xSize * ySize));
    hipMemcpy(image8bits, image8bitsCOPY, sizeof(unsigned char) * xSize * ySize, hipMemcpyDefault); 
    millisec t = std::chrono::system_clock::now() - before;

    double ret = t.count();
    gpu_sobel8<<<gridSize, blockSize>>>(image8bitsCOPY, xSize, ySize);
    hipDeviceSynchronize();
    hipFree(image8bitsCOPY);
    return ret;
}

double    GpuSobelMemcopyNotCount::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);

    auto before = system_clock::now();
    short *image16bitsCOPY;
    hipMallocManaged(&image16bitsCOPY, sizeof(short) * (xSize * ySize));
    hipMemcpy(image16bits, image16bitsCOPY, sizeof(short) * xSize * ySize, hipMemcpyDefault); 
    millisec t = std::chrono::system_clock::now() - before;
    double ret = t.count();

    gpu_sobel16<<<gridSize, blockSize>>>(image16bitsCOPY, xSize, ySize);

    hipDeviceSynchronize();
    hipFree(image16bitsCOPY);
    return ret;
}