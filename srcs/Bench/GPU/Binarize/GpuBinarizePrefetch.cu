#include "GpuBinarizePrefetch.hpp"

GpuBinarizePrefetch::GpuBinarizePrefetch(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuBinarizePrefetch::~GpuBinarizePrefetch()  
{
}


double    GpuBinarizePrefetch::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    int device = -1;
    hipGetDevice(&device);

    hipMemPrefetchAsync(image8bits, xSize * ySize * sizeof(unsigned char), device, NULL);
    gpu_binarise8<<<gridSize, blockSize>>>(image8bits, xSize, ySize, (unsigned char)BINARIZE_SEUIL, (unsigned char)BINARIZE_SEUIL_BAS, (unsigned char)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    return 0;
}

double    GpuBinarizePrefetch::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(image16bits, xSize * ySize * sizeof(short), device, NULL);
    gpu_binarise16<<<gridSize, blockSize>>>(image16bits, xSize, ySize, (short)BINARIZE_SEUIL, (short)BINARIZE_SEUIL_BAS, (short)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    return 0;
}