#include "GpuBinarizeCopy.hpp"

GpuBinarizeCopy::GpuBinarizeCopy(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuBinarizeCopy::~GpuBinarizeCopy()  
{
}

double    GpuBinarizeCopy::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    unsigned char *image8bitsCOPY;
    hipMallocManaged(&image8bitsCOPY, sizeof(unsigned char) * (xSize * ySize));

    gpu_binarise_copy8<<<gridSize, blockSize>>>(image8bits, image8bitsCOPY, xSize, ySize, (unsigned char)BINARIZE_SEUIL, (unsigned char)BINARIZE_SEUIL_BAS, (unsigned char)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    hipFree(image8bitsCOPY);
    return 0;
}


double    GpuBinarizeCopy::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    short *image16bitsCOPY;
    hipMallocManaged(&image16bitsCOPY, sizeof(short) * (xSize * ySize));

    gpu_binarise_copy16<<<gridSize, blockSize>>>(image16bits, image16bitsCOPY, xSize, ySize, (short)BINARIZE_SEUIL, (short)BINARIZE_SEUIL_BAS, (short)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    hipFree(image16bitsCOPY);
    return 0;
}


