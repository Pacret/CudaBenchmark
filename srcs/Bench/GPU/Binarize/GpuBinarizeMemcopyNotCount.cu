#include "GpuBinarizeMemcopyNotCount.hpp"

GpuBinarizeMemcopyNotCount::GpuBinarizeMemcopyNotCount(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuBinarizeMemcopyNotCount::~GpuBinarizeMemcopyNotCount()  
{
}


double    GpuBinarizeMemcopyNotCount::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    

    auto before = system_clock::now();
    unsigned char *image8bitsCOPY;
    
    hipMallocManaged(&image8bitsCOPY, sizeof(unsigned char) * (xSize * ySize));
    hipMemcpy(image8bits, image8bitsCOPY, sizeof(unsigned char) * xSize * ySize, hipMemcpyDefault); 
    millisec t = std::chrono::system_clock::now() - before;

    double ret = t.count();
    gpu_binarise8<<<gridSize, blockSize>>>(image8bitsCOPY, xSize, ySize, (unsigned char)BINARIZE_SEUIL, (unsigned char)BINARIZE_SEUIL_BAS, (unsigned char)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    hipFree(image8bitsCOPY);
    return ret;
}

double    GpuBinarizeMemcopyNotCount::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);

    auto before = system_clock::now();
    short *image16bitsCOPY;
    hipMallocManaged(&image16bitsCOPY, sizeof(short) * (xSize * ySize));
    hipMemcpy(image16bits, image16bitsCOPY, sizeof(short) * xSize * ySize, hipMemcpyDefault); 
    millisec t = std::chrono::system_clock::now() - before;
    double ret = t.count();


    gpu_binarise16<<<gridSize, blockSize>>>(image16bitsCOPY, xSize, ySize, (short)BINARIZE_SEUIL, (short)BINARIZE_SEUIL_BAS, (short)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    hipFree(image16bitsCOPY);
    return ret;
}