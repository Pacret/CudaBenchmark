#include "GpuBinarizeMemcopy.hpp"

GpuBinarizeMemcopy::GpuBinarizeMemcopy(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuBinarizeMemcopy::~GpuBinarizeMemcopy()  
{
}


double    GpuBinarizeMemcopy::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    unsigned char *image8bitsCOPY;
    hipMallocManaged(&image8bitsCOPY, sizeof(unsigned char) * (xSize * ySize));

    hipMemcpyAsync(image8bits, image8bitsCOPY, sizeof(unsigned char) * xSize * ySize, hipMemcpyDefault); 
    
    gpu_binarise8<<<gridSize, blockSize>>>(image8bitsCOPY, xSize, ySize, (unsigned char)BINARIZE_SEUIL, (unsigned char)BINARIZE_SEUIL_BAS, (unsigned char)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    hipFree(image8bitsCOPY);
    return 0;
}

double    GpuBinarizeMemcopy::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);

    short *image16bitsCOPY;
    hipMallocManaged(&image16bitsCOPY, sizeof(short) * (xSize * ySize));

    hipMemcpy(image16bits, image16bitsCOPY, sizeof(short) * xSize * ySize, hipMemcpyDefault); 

    gpu_binarise16<<<gridSize, blockSize>>>(image16bitsCOPY, xSize, ySize, (short)BINARIZE_SEUIL, (short)BINARIZE_SEUIL_BAS, (short)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    hipFree(image16bitsCOPY);
    return 0;
}