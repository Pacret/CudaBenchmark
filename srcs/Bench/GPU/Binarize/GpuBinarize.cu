#include "GpuBinarize.hpp"

GpuBinarize::GpuBinarize(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

GpuBinarize::~GpuBinarize()  
{
}

double    GpuBinarize::process8bits()
{
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    gpu_binarise8<<<gridSize, blockSize>>>(image8bits, xSize, ySize, (unsigned char)BINARIZE_SEUIL, (unsigned char)BINARIZE_SEUIL_BAS, (unsigned char)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    return 0;
}


double    GpuBinarize::process16bits()
{    
    dim3 blockSize(16, 16);
    dim3 gridSize((xSize + blockSize.x - 1) / blockSize.x, (ySize + blockSize.y - 1) / blockSize.y);
    
    gpu_binarise16<<<gridSize, blockSize>>>(image16bits, xSize, ySize, (short)BINARIZE_SEUIL, (short)BINARIZE_SEUIL_BAS, (short)BINARIZE_SEUIL_HAUT);
    hipDeviceSynchronize();
    return 0;
}


