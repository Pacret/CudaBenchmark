#include "hip/hip_runtime.h"
#include "CpuSobel.hpp"

CpuSobel::CpuSobel(std::string name, int xSize, int ySize) : Bench(name, xSize, ySize)
{
}

CpuSobel::~CpuSobel()  
{
}

void cpu_sobel8(
    unsigned char* image,
    int xSize,
    int ySize)
	// -------------------------------------------------------------------
	//	Filtre de Sobel sur Fenetre
	//	Recherche de contours horizontaux & verticaux
	//		-1	-2	-1				-1	0	+1
	//		0	0	0		puis	-2	0	+2
	//		+1	+2	+1				-1	0	+1
	// -------------------------------------------------------------------
	//	28/05/02	JPE : Creation
	//	09/04/03	JPE : version UNIQUE
	// -------------------------------------------------------------------
{
	long mgrisH = 0;
	long mgrisV = 0;
	long mgris = 0;
	long x=0,y=0;
	unsigned char *tabligneprec = NULL;
	unsigned char *tablignecour = NULL;
	unsigned char *adrtmp = NULL;
	unsigned char *adrLine = NULL;
	unsigned char *lpPix = NULL;

    long m_xg, m_yh, m_xd, m_yb; 
    m_xg = m_yh = 0;
    m_xd = xSize;
    m_yb = ySize;
    
    // Init des lignes :
	tabligneprec = new unsigned char[xSize];
	tablignecour = new unsigned char[xSize];
    
    adrLine = image + (m_yh*ySize);
    lpPix = adrLine + m_xg;
	for(x=m_xg;x<=m_xd;x++){
	    tabligneprec[x] = (unsigned char)*(lpPix);
		lpPix = lpPix + 1;
	}

    // les lignes constituant la fenetre ne sont pas prises en compte :
	m_xg++; m_yh++; m_xd--; m_yb--;
	// Parcours de la Fenetre :
	for(y=m_yh;y<m_yb;y++){
        // Ligne courante :
		m_xg--; m_xd++;
		lpPix = adrLine + m_xg;
		for(x=m_xg;x<m_xd;x++){
		    tablignecour[x] = (unsigned char)*(lpPix);
			lpPix = lpPix + 1;
		}
		m_xg++; m_xd--;
		// Parcours de la ligne courante :
		lpPix = adrLine + (m_xg);
		for(x=m_xg;x<m_xd;x++){

		    // Sobel Horizontal :
			mgrisH = 
			    (unsigned char)*(image + ((y+1)*ySize) + (x-1))
				+ (2*(unsigned char)*(image + ((y+1)*ySize) + x))
				+ (unsigned char)*(image + ((y+1)*ySize) + (x+1))
				- tabligneprec[x-1] - (2*tabligneprec[x]) - tabligneprec[x+1];
			// Sobel Vertical :
			mgrisV = 
			    tabligneprec[x+1] - tabligneprec[x-1]
				+ (2*tablignecour[x+1]) - (2*tablignecour[x-1])
				+ (unsigned char)*(image + ((y+1)*ySize) + (x+1))
				- (unsigned char)*(image + ((y+1)*ySize) + (x-1));
			// Sobel : Somme des Valeurs Absolues
			mgris = abs(mgrisH) + abs(mgrisV);
			// Test :
			if(mgris>SOBEL_GREY_VALUE)
                *lpPix = (unsigned char)SOBEL_GREY_VALUE;
			else
                *lpPix = (unsigned char)(mgris);
			lpPix = lpPix + 1;
        }
		// Ligne suivante :
		adrtmp = tabligneprec;
		tabligneprec = tablignecour;
		tablignecour = adrtmp;
		adrLine = adrLine + ySize;
	}
	// Liberation :
	if (tabligneprec!=NULL) delete[] tabligneprec;
	if (tablignecour!=NULL) delete[] tablignecour;
}

double    CpuSobel::process8bits()
{
    cpu_sobel8(image8bits, xSize, ySize);
    return 0;
}


void cpu_sobel16(
    short* image,
    int xSize,
    int ySize)
	// -------------------------------------------------------------------
	//	Filtre de Sobel sur Fenetre
	//	Recherche de contours horizontaux & verticaux
	//		-1	-2	-1				-1	0	+1
	//		0	0	0		puis	-2	0	+2
	//		+1	+2	+1				-1	0	+1
	// -------------------------------------------------------------------
	//	28/05/02	JPE : Creation
	//	09/04/03	JPE : version UNIQUE
	// -------------------------------------------------------------------
{
	long mgrisH = 0;
	long mgrisV = 0;
	long mgris = 0;
	long x=0,y=0;
	short *tabligneprec = NULL;
	short *tablignecour = NULL;
	short *adrtmp = NULL;
	short *adrLine = NULL;
	short *lpPix = NULL;

    long m_xg, m_yh, m_xd, m_yb; 
    m_xg = m_yh = 0;
    m_xd = xSize;
    m_yb = ySize;
    
    // Init des lignes :
	tabligneprec = new short[xSize];
	tablignecour = new short[xSize];
    
    adrLine = image + (m_yh*ySize);
    lpPix = adrLine + m_xg;
	for(x=m_xg;x<=m_xd;x++){
	    tabligneprec[x] = (short)*(lpPix);
		lpPix = lpPix + 1;
	}

    // les lignes constituant la fenetre ne sont pas prises en compte :
	m_xg++; m_yh++; m_xd--; m_yb--;
	// Parcours de la Fenetre :
	for(y=m_yh;y<m_yb;y++){
        // Ligne courante :
		m_xg--; m_xd++;
		lpPix = adrLine + m_xg;
		for(x=m_xg;x<=m_xd;x++){

		    tablignecour[x] = (short)*(lpPix);
			lpPix = lpPix + 1;
		}
		m_xg++; m_xd--;
		// Parcours de la ligne courante :
		lpPix = adrLine + (m_xg);
		for(x=m_xg;x<m_xd;x++){

            // std::cout << "process (" << x << ", " << y << ")" << std::endl; 
		    // Sobel Horizontal :
			mgrisH = 
			    (short)*(image + ((y+1)*ySize) + (x-1))
				+ (2*(short)*(image + ((y+1)*ySize) + x))
				+ (short)*(image + ((y+1)*ySize) + (x+1))
				- tabligneprec[x-1] - (2*tabligneprec[x]) - tabligneprec[x+1];
			// Sobel Vertical :
			mgrisV = 
			    tabligneprec[x+1] - tabligneprec[x-1]
				+ (2*tablignecour[x+1]) - (2*tablignecour[x-1])
				+ (short)*(image + ((y+1)*ySize) + (x+1))
				- (short)*(image + ((y+1)*ySize) + (x-1));
			// Sobel : Somme des Valeurs Absolues
			mgris = abs(mgrisH) + abs(mgrisV);
			// Test :
			if(mgris>100)
                *lpPix = 100;
			else
                *lpPix = (short)(mgris);
			lpPix = lpPix + 1;
        }
		// Ligne suivante :
		adrtmp = tabligneprec;
		tabligneprec = tablignecour;
		tablignecour = adrtmp;
		adrLine = adrLine + ySize;
	}
	// Liberation :
	if (tabligneprec!=NULL) delete[] tabligneprec;
	if (tablignecour!=NULL) delete[] tablignecour;
}

double    CpuSobel::process16bits()
{
    cpu_sobel16(image16bits, xSize, ySize);
    return 0;
}


