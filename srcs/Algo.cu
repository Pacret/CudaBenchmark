
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpu_binarise_copy8(unsigned char* imagecpu, unsigned char* imagegpu, int xSize, int ySize, unsigned char seuil, unsigned char valbas, unsigned char valhaut) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        unsigned char pixel = imagecpu[idx];
        imagegpu[idx] = (pixel <= seuil) ? valbas : valhaut;
    }
}

__global__ void gpu_binarise_copy16(short* imagecpu, short *imagegpu, int xSize, int ySize, short seuil, short valbas, short valhaut) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        short pixel = imagecpu[idx];
        imagegpu[idx] = (pixel <= seuil) ? valbas : valhaut;
    }
}
 


__global__ void gpu_binarise8(unsigned char* image, int xSize, int ySize, unsigned char seuil, unsigned char valbas, unsigned char valhaut) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        unsigned char pixel = image[idx];
        image[idx] = (pixel <= seuil) ? valbas : valhaut;
    }
}

__global__ void gpu_binarise16(short* image, int xSize, int ySize, short seuil, short valbas, short valhaut) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < xSize && y < ySize) {
        int idx = y * xSize + x;
        short pixel = image[idx];
        image[idx] = (pixel <= seuil) ? valbas : valhaut;
    }
}


__global__ void gpu_sobel8(unsigned char* image, int xSize, int ySize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	long mgrisH = 0;
	long mgrisV = 0;
	long mgris = 0;

    if (x > 0 && x < xSize - 1 && y > 0 && y < ySize - 1) {
        int idx = y * xSize + x;
        mgrisH = (unsigned char)*(image + ((y+1)*ySize) + (x-1))
				+ (2*(unsigned char)*(image + ((y+1)*ySize) + x))
				+ (unsigned char)*(image + ((y+1)*ySize) + (x+1))
                - (unsigned char)*(image + ((y-1)*ySize) + (x-1))
				- (2*(unsigned char)*(image + ((y-1)*ySize) + x))
				- (unsigned char)*(image + ((y-1)*ySize) + (x+1));
        mgrisV = (unsigned char)*(image + ((y-1)*ySize) + (x+1))
                - (unsigned char)*(image + ((y-1)*ySize) + (x-1))
                + (2*(unsigned char)*(image + (y*ySize) + (x+1)))
                - (2*(unsigned char)*(image + (y*ySize) + (x-1)))
				+ (unsigned char)*(image + ((y+1)*ySize) + (x+1))
				- (unsigned char)*(image + ((y+1)*ySize) + (x-1));
        mgris = abs(mgrisH) + abs(mgrisV);
        if(mgris>100)
            image[idx] = (unsigned char)100;
		else
            image[idx] = (unsigned char)(mgris);
    }
}

__global__ void gpu_sobel16(short* image, int xSize, int ySize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

	long mgrisH = 0;
	long mgrisV = 0;
	long mgris = 0;

    if (x > 0 && x < xSize - 1 && y > 0 && y < ySize - 1) {
        int idx = y * xSize + x;
        mgrisH = (short)*(image + ((y+1)*ySize) + (x-1))
				+ (2*(short)*(image + ((y+1)*ySize) + x))
				+ (short)*(image + ((y+1)*ySize) + (x+1))
                - (short)*(image + ((y-1)*ySize) + (x-1))
				- (2*(short)*(image + ((y-1)*ySize) + x))
				- (short)*(image + ((y-1)*ySize) + (x+1));
        mgrisV = (short)*(image + ((y-1)*ySize) + (x+1))
                - (short)*(image + ((y-1)*ySize) + (x-1))
                + (2*(short)*(image + (y*ySize) + (x+1)))
                - (2*(short)*(image + (y*ySize) + (x-1)))
				+ (short)*(image + ((y+1)*ySize) + (x+1))
				- (short)*(image + ((y+1)*ySize) + (x-1));
        mgris = abs(mgrisH) + abs(mgrisV);
        if(mgris>100)
            image[idx] = (short)100;
		else
            image[idx] = (short)(mgris);
    }
}
